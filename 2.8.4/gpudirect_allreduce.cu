#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
#include "nccl.h"
#include "mpi.h"
#include <sys/time.h>


#define MPICHECK(cmd) do {                                  \
    int e = cmd;                                            \
    if (e != MPI_SUCCESS) {                                 \
        printf("Failed: MPI error %s:%d '%d'\n",            \
                __FILE__, __LINE__, e);                     \
    }                                                       \
} while (0)


#define CUDACHECK(cmd) do {                                 \
    hipError_t e = cmd;                                    \
    if (e != hipSuccess) {                                 \
        printf("Failed: CUDA error %s:%d '%s'\n",           \
                __FILE__, __LINE__, hipGetErrorString(e)); \
    }                                                       \
} while (0)


#define NCCLCHECK(cmd) do {                                 \
    ncclResult_t r = cmd;                                   \
    if (r != ncclSuccess) {                                 \
        printf("Failed: NCCL error %s:%d '%s'\n",           \
                __FILE__, __LINE__, ncclGetErrorString(r)); \
    }                                                       \
} while (0)


double get_elapsed_time(struct timeval *begin, struct timeval *end)
{
    return (end->tv_sec - begin->tv_sec) * 1000000
            + (end->tv_usec - begin->tv_usec);
}


int main(int argc, char* argv[]) {
    int size = 32*1024*1024;
    if (argc>=2) {
       size = atoi(argv[1])*1024;
    }

    struct timeval start, end;
    int myRank, nRanks;
    long localRank;

    localRank = strtol(getenv("OMPI_COMM_WORLD_LOCAL_RANK"), NULL, 10);
    CUDACHECK(hipSetDevice(localRank));

    MPICHECK(MPI_Init(&argc, &argv));
    MPICHECK(MPI_Comm_rank(MPI_COMM_WORLD, &myRank));
    MPICHECK(MPI_Comm_size(MPI_COMM_WORLD, &nRanks));


    ncclUniqueId id;
    ncclComm_t comm;
    float *sendbuff, *recvbuff;
    hipStream_t s;

    if (myRank == 0) ncclGetUniqueId(&id);
    MPICHECK(MPI_Bcast((void*)&id, sizeof(id), MPI_BYTE, 0, MPI_COMM_WORLD));


    CUDACHECK(hipMalloc(&sendbuff, size * sizeof(float)));
    CUDACHECK(hipMalloc(&recvbuff, size * sizeof(float)));
    CUDACHECK(hipStreamCreate(&s));


    NCCLCHECK(ncclCommInitRank(&comm, nRanks, id, myRank));

    double sum=0.0;
    for (int i=0; i<100; i++) {

    CUDACHECK(hipStreamSynchronize(s));
    gettimeofday(&start, NULL);


    NCCLCHECK(ncclAllReduce((const void*)sendbuff, (void*)recvbuff, size, ncclFloat, ncclSum, comm, s));


    CUDACHECK(hipStreamSynchronize(s));
    gettimeofday(&end, NULL);

    {
	double us;
      	us = get_elapsed_time(&start, &end);
	sum += us/1000.0;
    }

    }

    printf("(Rank %d) time: %.3lf ms\n", myRank, sum/100);

    CUDACHECK(hipFree(sendbuff));
    CUDACHECK(hipFree(recvbuff));


    ncclCommDestroy(comm);


    MPICHECK(MPI_Finalize());


    // printf("[MPI Rank %d] Succcess\n", myRank);
    return 0;
}